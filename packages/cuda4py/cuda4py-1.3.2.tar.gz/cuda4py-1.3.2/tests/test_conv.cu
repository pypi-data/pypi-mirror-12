#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
/// Position convolution kernel center at (0, 0) in the image
////////////////////////////////////////////////////////////////////////////////
extern "C"
__global__ void padKernel(
    dtype *d_Dst,
    dtype *d_Src,
    int fftH,
    int fftW,
    int kernelH,
    int kernelW,
    int kernelY,
    int kernelX
)
{
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    const int x = blockDim.x * blockIdx.x + threadIdx.x;

    if (y < kernelH && x < kernelW)
    {
        int ky = y - kernelY;

        if (ky < 0)
        {
            ky += fftH;
        }

        int kx = x - kernelX;

        if (kx < 0)
        {
            kx += fftW;
        }

        d_Dst[ky * fftW + kx] = d_Src[y * kernelW + x];
    }
}


////////////////////////////////////////////////////////////////////////////////
// Prepare data for "pad to border" addressing mode
////////////////////////////////////////////////////////////////////////////////
extern "C"
__global__ void padData(
    dtype *d_Dst,
    dtype *d_Src,
    int fftH,
    int fftW,
    int dataH,
    int dataW,
    int kernelH,
    int kernelW,
    int kernelY,
    int kernelX
)
{
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int borderH = dataH + kernelY;
    const int borderW = dataW + kernelX;

    if (y < fftH && x < fftW)
    {
        int dy, dx;

        if (y < dataH)
        {
            dy = y;
        }

        if (x < dataW)
        {
            dx = x;
        }

        if (y >= dataH && y < borderH)
        {
            dy = dataH - 1;
        }

        if (x >= dataW && x < borderW)
        {
            dx = dataW - 1;
        }

        if (y >= borderH)
        {
            dy = 0;
        }

        if (x >= borderW)
        {
            dx = 0;
        }

        d_Dst[y * fftW + x] = d_Src[dy * dataW + dx];
    }
}


// a = (a * b) * c
extern "C"
__global__ void mul_scale(dtype *a, const dtype *b, const dtype c, const int limit) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if (idx < limit) {
    idx <<= 1;
    dtype ax = a[idx], ay = a[idx + 1];
    dtype bx = b[idx], by = b[idx + 1];
    a[idx] = (ax * bx - ay * by) * c;
    a[idx + 1] = (ay * bx + ax * by) * c;
  }
}
